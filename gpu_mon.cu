
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <algorithm>
#include <sstream>
#include <chrono>
#include <thread>
#include <nvml.h>
#pragma comment(lib, "nvml")

#define sleep(t) std::this_thread::sleep_for(std::chrono::milliseconds(t))

// const float bytes_per_gb = (1 << 30);
const float bytes_per_gb = (1 << 30);
const float bytes_per_mib = (1 << 20);
const float ms_per_hour = 1000 * 3600;
const int max_grid_dim = (1 << 15);
const int max_block_dim = 1024;
const int max_sleep_time = 1e3;
const float sleep_interval = 1e16;
const int max_gpu_num = 32;

__global__ void default_script_kernel(char* array, size_t occupy_size) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= occupy_size) return;
  array[i]++;
}

void launch_default_script(char** array, size_t occupy_size,
                           std::vector<int>& grid_dim,
                           std::vector<int>& gpu_ids) {
  int gd = std::min(grid_dim[rand() % grid_dim.size()],
                    int(occupy_size / max_block_dim));
  for (int id : gpu_ids) {
    hipSetDevice(id);
    default_script_kernel<<<gd, max_block_dim, 0, NULL>>>(array[id],
                                                          occupy_size);
  }
}

void run_default_script(char** array, size_t occupy_size, float total_time,
                        std::vector<int>& gpu_ids) {
  printf("Running default script >>>>>>>>>>>>>>>>>>>>\n");
  for (int id : gpu_ids) {
    hipSetDevice(id);
    // cudaDeviceSynchronize();
    hipError_t status = hipMalloc(&array[id], occupy_size);
  }
  srand(time(NULL));
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time;
  size_t cnt = 0, sleep_time;
  std::vector<int> grid_dim;
  for (int i = 1; i <= max_grid_dim; i <<= 1) {
    grid_dim.push_back(i);
  }
  hipEventRecord(start, 0);
  while (true) {
    launch_default_script(array, occupy_size, grid_dim, gpu_ids);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    if (time / ms_per_hour > total_time) break;
    if (!((++cnt) % size_t(sleep_interval / occupy_size))) {
      cnt = 0;
      printf("Occupied time: %.2f hours\n", time / ms_per_hour);
      sleep_time = rand() % max_sleep_time + 1;
      sleep(sleep_time);
    }
  }
  hipEventDestroy(start);
  hipEventDestroy(stop);
  for (int id : gpu_ids) {
    hipFree(array[id]);
  }
}

void process_args(int argc, char** argv, size_t& occupy_size, float& total_time,
                  std::vector<int>& gpu_ids, std::string& script_path) {
  if (argc < 4) {
    printf(
        "Arguments: <**any dummy augments**>  <GPU Memory (MiB)> <Occupied Time (h)> <GPU ID> <OPTIONAL: "
        "Script Path>\n");
    throw std::invalid_argument("Invalid argument number");
  }
  int gpu_num;
  hipGetDeviceCount(&gpu_num);
  int id;
  std::string s(argv[3]);
  std::replace(s.begin(), s.end(), ',', ' ');
  std::stringstream ss;
  ss << s;
  while (ss >> id) {
    gpu_ids.push_back(id);
  }
  if (gpu_ids.size() == 1 && gpu_ids[0] == -1) {
    gpu_ids[0] = 0;
    for (int i = 1; i < gpu_num; ++i) {
      gpu_ids.push_back(i);
    }
  }
  for (int i : gpu_ids) {
    if (i < 0 || i >= gpu_num) {
      printf("Invalid GPU ID (%d GPU in total): %d\n", i, gpu_num);
      throw std::invalid_argument("Invalid GPU ID");
    }
  }

  float occupy_mem;
  size_t total_size, avail_size;
  hipMemGetInfo(&avail_size, &total_size);
  sscanf(argv[1], "%f", &occupy_mem);
  sscanf(argv[2], "%f", &total_time);
  if (occupy_mem <= 0) {
    printf("GPU memory must be positive: %.2f\n", occupy_mem);
    throw std::invalid_argument("Invalid GPU memory");
  }
  if (total_time < 0) {
    printf("Occupied time must be positive: %.2f\n", total_time);
    throw std::invalid_argument("Invalid occupied time");
  }
  occupy_size = occupy_mem * bytes_per_mib;
  if (occupy_size > total_size) {
    printf("GPU memory exceeds maximum (%.2f MiB): %.2f\n",
           total_size / bytes_per_mib, occupy_mem);
    throw std::invalid_argument("Exceed maximal GPU memory");
  }

  printf("GPU memory (MiB): %.2f\n", occupy_mem);
  printf("Occupied time (h): %.2f\n", total_time);
  if (argc == 4) {
    printf("GPU ID: ");
    for (int id = 0; id < gpu_ids.size(); ++id) {
      printf("%d%c", gpu_ids[id], ",\n"[id == gpu_ids.size() - 1]);
    }
  } else {
    script_path = argv[4];
    printf("Script path: %s\n", script_path.c_str());
  }

}

void allocate_mem(char** array, size_t occupy_size, std::vector<int>& gpu_ids) {
  std::vector<bool> allocated(max_gpu_num, false);
  int cnt = 0;
  
  while (true) {
    printf("Clean & Try allocate GPU memory %d times >>>>>>>>>>>>>>>>>>>>\n", ++cnt);
    hipDeviceReset();
    bool all_allocated = true;
    for (int id : gpu_ids) {
      if (!allocated[id]) {
        nvmlDevice_t device;
        nvmlReturn_t device_handle = nvmlDeviceGetHandleByIndex(id, &device);
        // cudaSetDevice(id);
        // cudaError_t status = cudaMalloc(&array[id], occupy_size);
        nvmlMemory_t memory;
        nvmlReturn_t memory_return = nvmlDeviceGetMemoryInfo(device, &memory);

        // unsigned long long total_size = memory.total;
        // unsigned long long used_size = memory.used;
        unsigned long long free_size = memory.free;
        // cudaMemGetInfo(&avail_size, &total_size);
        // if (status != cudaSuccess) 
        if (occupy_size > free_size)
        {
          printf(
              "GPU-%d: Failed to allocate %.2f MiB GPU memory (%.2f MiB "
              "available)\n",
              id, occupy_size / bytes_per_mib, free_size / bytes_per_mib);
          all_allocated = false;
          // break;
        } 
        else 
        {
          // if (status == cudaSuccess)
          // {
          //   allocated[id] = true;
          //   printf(
          //       "GPU-%d: Successfully allocate %.2f GB GPU memory (%.2f GB "
          //       "available)\n",
          //       id, occupy_size / bytes_per_gb, free_size / bytes_per_gb);
          // } 
          // else 
          // {
          //   printf("GPU-%d: Error %d\n", id, status);
          //   all_allocated = false;
          // }
          allocated[id] = true;
          printf(
              // "GPU-%d: Successfully allocate %.2f GB GPU memory (%.2f GB "
              // "available)\n",
              // id, occupy_size / bytes_per_gb, free_size / bytes_per_gb

              "GPU-%d: Successfully allocate %.2f MiB GPU memory (%.2f MiB "
              "available)\n",
              id, occupy_size / bytes_per_mib, free_size / bytes_per_mib
              );
        }
      }
    }
    
    if (all_allocated) break;
    sleep(5000);
  }
  printf("Successfully allocate memory on all GPUs!\n");
}

void run_custom_script(char** array, std::vector<int>& gpu_ids,
                       std::string script_path) {
  // std::cin.get();
  printf("Running custom script >>>>>>>>>>>>>>>>>>>>\n");
  // cudaDeviceReset();
  // std::cin.get();
  // for (int id : gpu_ids) {
  //   cudaFree(array[id]);
  // }
  // // cudaDeviceReset();
  // std::cin.get();
  nvmlShutdown();
  // std::cin.get();
  std::string cmd = "sh " + script_path;
  std::system(cmd.c_str());
}

bool has_suffix(const std::string &str, const std::string &suffix)
{
    return str.size() >= suffix.size() &&
           str.compare(str.size() - suffix.size(), suffix.size(), suffix) == 0;
}

int main(int argc, char** argv) {
  size_t occupy_size;
  float total_time;
  std::vector<int> gpu_ids;
  std::string script_path;
  char* array[max_gpu_num];
  bool run_custom = false;

  int real_argc = argc;
  char** real_argv = argv;

  // Process arguments
  if (argc < 4) {
    printf(
        "Arguments: <**any dummy augments**>  <GPU Memory (MiB)> <Occupied Time (h)> <GPU ID> <OPTIONAL: "
        "Script Path>\n");
    printf("Example 1: Occupy 16000 MB GPU memory for 24 hours using GPU 0, 1, 2, 3 to run default script.\n");
    printf("./gpu_mon 0 16000 24 0,1,2,3\n");
    printf("Example 2: Occupy 16000 MB GPU memory for 24 hours using GPU 0, 1, 2, 3 to run custom script `run.sh`.\n");
    printf("./gpu_mon 0 16000 24 0,1,2,3 run.sh\n");
    printf("Example 3: Occupy 16000 MB GPU memory for 24 hours using GPU 0, 1, 2, 3 to run default script, with some dummy augrments\n");
    printf("./gpu_mon dummy_arg1 dummy_arg2 0 16000 24 0,1,2,3\n");
    throw std::invalid_argument("Invalid argument number");
  } else {
    // if the last argv is end with .sh
    if (has_suffix(argv[argc-1], ".sh"))
    {
      // run custom script
      run_custom = true;
      real_argc = 5;
    } else {
      // run default script
      run_custom = false;
      real_argc = 4;
    }

    // remove dummy augments in real_argv and only keep the the first, and last number of real_argc augments
    real_argv = new char*[real_argc];
    real_argv[0] = argv[0]; // filename
    for (int i = 1; i < real_argc; ++i)
    {
      real_argv[i] = argv[argc-real_argc+i];
    }
  }

  // print to check real_argc and real_argv
  printf("real_argc: %d\n", real_argc);
  for (int i = 0; i < real_argc; ++i)
  {
    printf("real_argv[%d]: %s\n", i, real_argv[i]);
  }

  nvmlReturn_t init_ptr;
  init_ptr = nvmlInit();


  if (init_ptr == NVML_SUCCESS){
    process_args(real_argc, real_argv, occupy_size, total_time, gpu_ids, script_path);
    hipDeviceReset();
    allocate_mem(array, occupy_size, gpu_ids);
  
    if (run_custom == false) {
      run_default_script(array, occupy_size, total_time, gpu_ids);
    } else {
      run_custom_script(array, gpu_ids, script_path);
    }

    // nvmlShutdown();

    return 0;
  }
}
